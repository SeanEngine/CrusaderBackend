//
// Created by Dylan on 6/2/2022.
//

#include "CudaAssert.cuh"

namespace seblas {
    void assertCuda(const char* file, int line){
        hipError_t error = hipGetLastError();
        if(error != hipSuccess){
            logFatal(seio::LOG_SEG_SEBLAS, string("Device error encountered:") + hipGetErrorString(error));
            logFatal(seio::LOG_SEG_SEBLAS, "line: " + to_string(line) + "  file: " + string(file));
            throw runtime_error("line: " + to_string(line) + "  file: " + string(file));
        }
    }
} // seblas
#include "hip/hip_runtime.h"

#include "cuConv.cuh"

#define BM 128
#define BN 128
#define BK 8
#define RM 8
#define RN 8

#define toFloat4R(ptr) (reinterpret_cast<float4*>(&(ptr))[0])

namespace seblas{
    /**
    * @brief The kernel for the convolution with 4D filters
    * @tparam BLOCK_M
    * @tparam BLOCK_N
    * @tparam BLOCK_K
    * @tparam REGIS_M
    * @tparam REGIS_N
    * @param A oc * ic * fh * fw
    * @param B on * ic * ih * iw
    * @param C on * oc * oh * ow
    * @param strideH
    * @param strideW
    * @param padH
    * @param padW
     *
     * [Unit test passed]
    */
    template<const int BLOCK_M, const int BLOCK_N, const int BLOCK_K,
            const int REGIS_M, const int REGIS_N>
    __global__ void gemmImplicit4D(Tensor* A, Tensor* B, Tensor* C, int strideH, int strideW, int padH, int padW,
            /*NULLABLE*/ Tensor* biases){
        
        // MatA: OC, IC * FH * FW; MatB: IC * FH * FW, OH * OW; Mat C: OC, OH * OW
        ///insert parameters
        const uint32 M = A->dims.n;
        const uint32 K = A->dims.c * A->dims.h * A->dims.w;
        const uint32 N = C->dims.n * C->dims.h * C->dims.w;
        
        const uint32 FH = A->dims.h;
        const uint32 FW = A->dims.w;
        const uint32 IC = B->dims.c;
        const uint32 IH = B->dims.h;
        const uint32 IW = B->dims.w;
        const uint32 OH = C->dims.h;
        const uint32 OW = C->dims.w;
        const uint32 OC = C->dims.c;
        
        ///allocate smems and registers
        //The shared memory tile
        __shared__ float tileA[2][BLOCK_K][BLOCK_M];  //transposed
        __shared__ float tileB[2][BLOCK_K][BLOCK_N];
        
        float regisA[2][REGIS_M];
        float regisB[2][REGIS_N];
        float regisC[REGIS_M][REGIS_N] = {0};
        
        const int threadDimX = BLOCK_N / REGIS_N;
        const int threadDimY = BLOCK_M / REGIS_M;
        const int threadCount = threadDimX * threadDimY;
        const int tid = threadIdx.y * threadDimX + threadIdx.x;
        
        ///register for buffering elements during transporting global to shared mem
        float bufferA[BLOCK_M * BLOCK_K / threadCount] = {0};
        float bufferB[BLOCK_N * BLOCK_K / threadCount] = {0};
        
        ///prepare configs for reading global
        float* ptrA = A->elements + blockIdx.y * BLOCK_M * K;
        float* ptrB = B->elements;
        const int blockM = blockIdx.y * BLOCK_M;
        const int blockN = blockIdx.x * BLOCK_N;
        
        const int readThreadPerRowA = BLOCK_K;
        const int readThreadPerRowB = BLOCK_N;
        
        //the location each thread should be reading relative to smem
        const int readRowA = tid / readThreadPerRowA;
        const int readColA = tid % readThreadPerRowA;
        
        const int readRowB = tid / readThreadPerRowB;
        const int readColB = tid % readThreadPerRowB;
        
        //these values are used to determine the amount of rows to jump
        //if there is the need to do read multiple times
        const int readRowStrideA = threadCount / readThreadPerRowA;
        const int readRowStrideB = threadCount / readThreadPerRowB;
        
        #pragma unroll
        for(int i=0; i<BLOCK_M; i+= readRowStrideA){
            if(blockM + readRowA + i < M && readColA < K){
                tileA[0][readColA][readRowA+i] = ptrA[(readRowA + i)*K + readColA];
            }else{
                tileA[0][readColA][readRowA+i] = 0;
            }
        }
        
        ///this section is modified from its original state to suit the need for implicit gemm
        ///we are using A special mapping to instantiate patches as trajectories of conv filters
        #pragma unroll
        for(int i=0; i<BLOCK_K; i+= readRowStrideB){
            if(readRowB + i< K && blockN + readColB < N){
                
                //map buffer matrix cords to the 3 dimensional feature cords
                int in = (readColB + blockN) / (OH * OW);
                int oh = ((readColB + blockN) % (OH * OW))/OW;
                int ow = ((readColB + blockN) % (OH * OW))%OW;
                int ic = (readRowB + i)/(FH * FW);
                int fh = ((readRowB + i)%(FH * FW))/FW;
                int fw = ((readRowB + i)%(FH * FW))%FW;
                int ih = oh * strideH - padH + fh;
                int iw = ow * strideW - padW + fw;
                //do memory access
                tileB[0][readRowB+i][readColB] = ih >= 0 && iw >= 0 && ih < IH && iw < IW ?
                                                 ptrB[in * IC * IH * IW + ic * IH * IW + ih * IW + iw] : 0;
            }else{
                tileB[0][readRowB+i][readColB] = 0;
            }
        }
        __syncthreads();
        
        
        #pragma unroll
        for(int rm = 0; rm < REGIS_M; rm += 4){
            toFloat4R(regisA[0][rm]) = toFloat4R(tileA[0][0][REGIS_M * threadIdx.y + rm]);
        }
        
        #pragma unroll
        for(int rn = 0; rn < REGIS_N; rn += 4){
            toFloat4R(regisB[0][rn]) = toFloat4R(tileB[0][0][REGIS_N * threadIdx.x + rn]);
        }
        
        ///main loop
        int writeStageFlag = 1;
        #pragma unroll
        for(int nextTileID = BLOCK_K; nextTileID < K + BLOCK_K; nextTileID+=BLOCK_K) {
            //prefetch
            if (nextTileID < K) {
                #pragma unroll
                for (int i = 0; i < BLOCK_M; i += readRowStrideA) {
                    int loadIndex = i / readRowStrideA;
                    bufferA[loadIndex] = blockM + readRowA + i < M && readColA + nextTileID < K ?
                                         ptrA[(readRowA + i) * K + readColA + nextTileID] : 0;
                }
                
                #pragma unroll
                for (int i = 0; i < BLOCK_K; i += readRowStrideB) {
                    
                    //calculate remapping
                    int loadIndex = i / readRowStrideB;
                    int in = (readColB + blockN) / (OH * OW);
                    int oh = ((readColB + blockN) % (OH * OW))/OW;
                    int ow = ((readColB + blockN) % (OH * OW))%OW;
                    int ic = (readRowB + i + nextTileID)/(FH * FW);
                    int fh = ((readRowB + i + nextTileID)%(FH * FW))/FW;
                    int fw = ((readRowB + i + nextTileID)%(FH * FW))%FW;
                    int ih = oh * strideH - padH + fh;
                    int iw = ow * strideW - padW + fw;
                    
                    //do memory access
                    bufferB[loadIndex] = (readRowB + i + nextTileID < K && blockN + readColB < N) && (ih >= 0 && iw >= 0)
                                         && (ih < IH && iw < IW)? ptrB[in * IC * IH * IW + ic * IH * IW + ih * IW + iw] : 0;
                }
            }
            
            int nextStageFlag = writeStageFlag ^ 1;
            
            //compute the part that is already in the registers and load the next segment
            #pragma unroll
            for (int i = 0; i < BLOCK_K - 1; i++) {
                
                #pragma unroll
                for (int rm = 0; rm < REGIS_M; rm += 4) {
                    toFloat4R(regisA[(i + 1) % 2][rm]) = toFloat4R(
                            tileA[nextStageFlag][i + 1][REGIS_M * threadIdx.y + rm]);
                }
                
                #pragma unroll
                for (int rn = 0; rn < REGIS_N; rn += 4) {
                    toFloat4R(regisB[(i + 1) % 2][rn]) = toFloat4R(
                            tileB[nextStageFlag][i + 1][REGIS_N * threadIdx.x + rn]);
                }
                
                #pragma unroll
                for (int rm = 0; rm < REGIS_M; rm++) {
                    #pragma unroll
                    for (int rn = 0; rn < REGIS_N; rn++) {
                        regisC[rm][rn] += regisA[i % 2][rm] * regisB[i % 2][rn];
                    }
                }
            }
            
            //load the data in the register buffers to tiles
            if (nextTileID < K) {
                #pragma unroll
                for (int i = 0; i < BLOCK_M; i += readRowStrideA) {
                    int loadIndex = i / readRowStrideA;
                    tileA[writeStageFlag][readColA][readRowA + i] = bufferA[loadIndex];
                }
                
                #pragma unroll
                for (int i = 0; i < BLOCK_K; i += readRowStrideB) {
                    int loadIndex = i / readRowStrideB;
                    tileB[writeStageFlag][readRowB + i][readColB] = bufferB[loadIndex];
                }
                
                __syncthreads();
                writeStageFlag ^= 1;  //switch
            }
            #pragma unroll
            for (int rm = 0; rm < REGIS_M; rm += 4) {
                toFloat4R(regisA[0][rm]) = toFloat4R(
                        tileA[nextStageFlag ^ 1][0][REGIS_M * threadIdx.y + rm]);
            }
            
            #pragma unroll
            for (int rn = 0; rn < REGIS_N; rn += 4) {
                toFloat4R(regisB[0][rn]) = toFloat4R(
                        tileB[nextStageFlag ^ 1][0][REGIS_N * threadIdx.x + rn]);
            }
            
            #pragma unroll
            for(int rm = 0; rm < REGIS_M; rm ++){
                #pragma unroll
                for(int rn = 0; rn < REGIS_N; rn ++){
                    regisC[rm][rn] += regisA[1][rm] * regisB[1][rn];
                }
            }
        }
        #pragma unroll
        for(int rm = 0; rm < REGIS_M; rm ++){
            #pragma unroll
            for(int rn = 0; rn < REGIS_N; rn ++){
                if((blockM + threadIdx.y * REGIS_M + rm < M && blockN + threadIdx.x * REGIS_N + rn < N)) {
                    float bias = biases == nullptr ? 0 : biases->elements[blockM + threadIdx.y * REGIS_M + rm];
                    //remapping is needed since the original output matrix would be ( OC, OH * OW * ON )
                    uint32 on = (blockN + threadIdx.x * REGIS_N + rn) / (OH * OW);
                    C->elements[on * OC * OH * OW + (blockM + threadIdx.y * REGIS_M + rm) * (OH * OW)
                                + (blockN + threadIdx.x * REGIS_N + rn)%(OH * OW)] = regisC[rm][rn] + bias;
                }
            }
        }
    }
    
    /**
     * @brief The back propagation of conv layers (relative to input features)
     * @tparam BLOCK_M
     * @tparam BLOCK_N
     * @tparam BLOCK_K
     * @tparam REGIS_M
     * @tparam REGIS_N
     * @param A OC * IC * FH * FW
     * @param B ON * OC * OH * OW
     * @param C ON * IC * IH * IW
     * @param strideH original input feature's strideH, but stands for upsampling factor
     * @param strideW original input feature's strideW, but stands for upsampling factor
     * @param newPadH pre-processed padding (different from original forward calculation)
     * @param newPadW pre-processed padding (different from original forward calculation)
     */
    template<const int BLOCK_M, const int BLOCK_N, const int BLOCK_K,
            const int REGIS_M, const int REGIS_N>
    __global__ void gemmImplicitBackprop(Tensor* A, Tensor* B, Tensor* C,
                 int strideH, int strideW, int newPadH, int newPadW){
        ///insert parameters
        //order changed for reversed convolution
        const uint32 M = A->dims.c;
        const uint32 K = A->dims.n * A->dims.h * A->dims.w;
        const uint32 SRC_K = A->dims.c * A->dims.h * A->dims.w;
        const uint32 N = C->dims.n * C->dims.h * C->dims.w;
        
        const uint32 FH = A->dims.h;
        const uint32 FW = A->dims.w;
        const uint32 IC = B->dims.c;
        const uint32 IH = B->dims.h;
        const uint32 IW = B->dims.w;
        const uint32 OH = C->dims.h;
        const uint32 OW = C->dims.w;
        const uint32 OC = C->dims.c;
        
        ///allocate smems and registers
        //The shared memory tile
        __shared__ float tileA[2][BLOCK_K][BLOCK_M];  //transposed
        __shared__ float tileB[2][BLOCK_K][BLOCK_N];
        
        float regisA[2][REGIS_M];
        float regisB[2][REGIS_N];
        float regisC[REGIS_M][REGIS_N] = {0};
        
        const int threadDimX = BLOCK_N / REGIS_N;
        const int threadDimY = BLOCK_M / REGIS_M;
        const int threadCount = threadDimX * threadDimY;
        const int tid = threadIdx.y * threadDimX + threadIdx.x;
        
        ///register for buffering elements during transporting global to shared mem
        float bufferA[BLOCK_M * BLOCK_K / threadCount] = {0};
        float bufferB[BLOCK_N * BLOCK_K / threadCount] = {0};
        
        ///prepare configs for reading global
        float* ptrA = A->elements;
        float* ptrB = B->elements;
        const int blockM = blockIdx.y * BLOCK_M;
        const int blockN = blockIdx.x * BLOCK_N;
        
        const int readThreadPerRowA = BLOCK_K;
        const int readThreadPerRowB = BLOCK_N;
        
        //the location each thread should be reading relative to smem
        const int readRowA = tid / readThreadPerRowA;
        const int readColA = tid % readThreadPerRowA;
        
        const int readRowB = tid / readThreadPerRowB;
        const int readColB = tid % readThreadPerRowB;
        
        //these values are used to determine the amount of rows to jump
        //if there is the need to do read multiple times
        const int readRowStrideA = threadCount / readThreadPerRowA;
        const int readRowStrideB = threadCount / readThreadPerRowB;
        
        #pragma unroll
        for(int i=0; i<BLOCK_M; i+= readRowStrideA){
            int rowFilterIndex = readColA / (FH * FW);
            int colFilterIndex = blockM + readRowA + i;
            int colIndex = (FH * FW - 1) - (readColA % (FH * FW));
            if(blockM + readRowA + i < M && readColA < K){
                //rotate 180 degrees
                tileA[0][readColA][readRowA+i] = ptrA[rowFilterIndex * SRC_K + colFilterIndex * (FH * FW) + colIndex];
            } else {
                tileA[0][readColA][readRowA+i] = 0;
            }
        }
        
        ///this section is modified from its original state to suit the need for implicit gemm
        ///we are using A special mapping to instantiate patches as trajectories of conv filters
        #pragma unroll
        for(int i=0; i<BLOCK_K; i+= readRowStrideB){
            if(readRowB + i< K && blockN + readColB < N){
                
                //map buffer matrix cords to the 3 dimensional feature cords
                int in = (readColB + blockN) / (OH * OW);
                int oh = ((readColB + blockN) % (OH * OW))/OW;
                int ow = ((readColB + blockN) % (OH * OW))%OW;
                int ic = (readRowB + i)/(FH * FW);
                int fh = ((readRowB + i)%(FH * FW))/FW;
                int fw = ((readRowB + i)%(FH * FW))%FW;
                int ih = oh - newPadH + fh;
                int iw = ow - newPadW + fw;
                int procIh = ih / strideH;
                int procIw = iw / strideW;
                //do memory access
                tileB[0][readRowB+i][readColB] = procIh >= 0 && procIw >= 0 && procIh < IH &&
                                                 procIw < IW && ih % strideH == 0 && iw % strideW == 0 ?
                                                 ptrB[in * IC * IH * IW + ic * IH * IW + procIh * IW + procIw] : 0;
            }
        }
        __syncthreads();
        
        
        #pragma unroll
        for(int rm = 0; rm < REGIS_M; rm += 4){
            toFloat4R(regisA[0][rm]) = toFloat4R(tileA[0][0][REGIS_M * threadIdx.y + rm]);
        }
        
        #pragma unroll
        for(int rn = 0; rn < REGIS_N; rn += 4){
            toFloat4R(regisB[0][rn]) = toFloat4R(tileB[0][0][REGIS_N * threadIdx.x + rn]);
        }
        
        ///main loop
        int writeStageFlag = 1;
        #pragma unroll
        for(int nextTileID = BLOCK_K; nextTileID < K + BLOCK_K; nextTileID+=BLOCK_K) {
            //prefetch
            if (nextTileID < K) {
                #pragma unroll
                for (int i = 0; i < BLOCK_M; i += readRowStrideA) {
                    int loadIndex = i / readRowStrideA;
                    int rowFilterIndex = (readColA + nextTileID) / (FH * FW);
                    int colFilterIndex = blockM + readRowA + i;
                    int colIndex = (FH * FW - 1) - ((readColA + nextTileID) % (FH * FW));
                    bufferA[loadIndex] = blockM + readRowA + i < M && readColA + nextTileID < K ?
                                         ptrA[rowFilterIndex * SRC_K + colFilterIndex * (FH * FW) + colIndex] : 0;
                }
                
                #pragma unroll
                for (int i = 0; i < BLOCK_K; i += readRowStrideB) {
                    
                    //calculate remapping
                    int loadIndex = i / readRowStrideB;
                    int in = (readColB + blockN) / (OH * OW);
                    int oh = ((readColB + blockN) % (OH * OW))/OW;
                    int ow = ((readColB + blockN) % (OH * OW))%OW;
                    int ic = (readRowB + i + nextTileID)/(FH * FW);
                    int fh = ((readRowB + i + nextTileID)%(FH * FW))/FW;
                    int fw = ((readRowB + i + nextTileID)%(FH * FW))%FW;
                    int ih = oh - newPadH + fh;
                    int iw = ow - newPadW + fw;
                    int procIh = ih / strideH;
                    int procIw = iw / strideW;
                    
                    //do memory access
                    bufferB[loadIndex] = (readRowB + i + nextTileID < K && blockN + readColB < N) && (procIh >= 0 && procIw >= 0)
                                         && (procIh < IH && procIw < IW) && ih % strideH == 0 && iw % strideW == 0 ?
                                         ptrB[in * IC * IH * IW + ic * IH * IW + procIh * IW + procIw] : 0;
                }
            }
            
            int nextStageFlag = writeStageFlag ^ 1;
            
            //compute the part that is already in the registers and load the next segment
            #pragma unroll
            for (int i = 0; i < BLOCK_K - 1; i++) {
                
                #pragma unroll
                for (int rm = 0; rm < REGIS_M; rm += 4) {
                    toFloat4R(regisA[(i + 1) % 2][rm]) = toFloat4R(
                            tileA[nextStageFlag][i + 1][REGIS_M * threadIdx.y + rm]);
                }
                
                #pragma unroll
                for (int rn = 0; rn < REGIS_N; rn += 4) {
                    toFloat4R(regisB[(i + 1) % 2][rn]) = toFloat4R(
                            tileB[nextStageFlag][i + 1][REGIS_N * threadIdx.x + rn]);
                }
                
                #pragma unroll
                for (int rm = 0; rm < REGIS_M; rm++) {
                    #pragma unroll
                    for (int rn = 0; rn < REGIS_N; rn++) {
                        regisC[rm][rn] += regisA[i % 2][rm] * regisB[i % 2][rn];
                    }
                }
            }
            
            //load the data in the register buffers to tiles
            if (nextTileID < K) {
                #pragma unroll
                for (int i = 0; i < BLOCK_M; i += readRowStrideA) {
                    int loadIndex = i / readRowStrideA;
                    tileA[writeStageFlag][readColA][readRowA + i] = bufferA[loadIndex];
                }
                
                #pragma unroll
                for (int i = 0; i < BLOCK_K; i += readRowStrideB) {
                    int loadIndex = i / readRowStrideB;
                    tileB[writeStageFlag][readRowB + i][readColB] = bufferB[loadIndex];
                }
                
                __syncthreads();
                writeStageFlag ^= 1;  //switch
            }
            #pragma unroll
            for (int rm = 0; rm < REGIS_M; rm += 4) {
                toFloat4R(regisA[0][rm]) = toFloat4R(
                        tileA[nextStageFlag ^ 1][0][REGIS_M * threadIdx.y + rm]);
            }
            
            #pragma unroll
            for (int rn = 0; rn < REGIS_N; rn += 4) {
                toFloat4R(regisB[0][rn]) = toFloat4R(
                        tileB[nextStageFlag ^ 1][0][REGIS_N * threadIdx.x + rn]);
            }
            
            #pragma unroll
            for(int rm = 0; rm < REGIS_M; rm ++){
                #pragma unroll
                for(int rn = 0; rn < REGIS_N; rn ++){
                    regisC[rm][rn] += regisA[1][rm] * regisB[1][rn];
                }
            }
        }
        #pragma unroll
        for(int rm = 0; rm < REGIS_M; rm ++){
            #pragma unroll
            for(int rn = 0; rn < REGIS_N; rn ++){
                if((blockM + threadIdx.y * REGIS_M + rm < M && blockN + threadIdx.x * REGIS_N + rn < N)) {
                    //remapping is needed since the original output matrix would be ( OC, OH * OW * ON )
                    uint32 on = (blockN + threadIdx.x * REGIS_N + rn) / (OH * OW);
                    C->elements[on * OC * OH * OW + (blockM + threadIdx.y * REGIS_M + rm) * (OH * OW)
                                + (blockN + threadIdx.x * REGIS_N + rn)%(OH * OW)] = regisC[rm][rn];
                }
            }
        }
    }
    
    ///[Unit Test Passed]
    template<const int BLOCK_M, const int BLOCK_N, const int BLOCK_K,
            const int REGIS_M, const int REGIS_N>
    __global__ void gemmImplicitError(Tensor* A, Tensor* B, Tensor* C, int strideH, int strideW, int padH, int padW){
        
        ///insert parameters
        const uint32 M = A->dims.c;
        const uint32 K = A->dims.h * A->dims.w * A->dims.n;
        const uint32 N = C->dims.c * C->dims.h * C->dims.w;

        const uint32 FH = A->dims.h;
        const uint32 FW = A->dims.w;
        const uint32 FC = A->dims.c;
        const uint32 IC = B->dims.c;
        const uint32 IH = B->dims.h;
        const uint32 IW = B->dims.w;
        const uint32 OH = C->dims.h;
        const uint32 OW = C->dims.w;

        ///allocate smems and registers
        //The shared memory tile
        __shared__ float tileA[2][BLOCK_K][BLOCK_M];  //transposed
        __shared__ float tileB[2][BLOCK_K][BLOCK_N];

        float regisA[2][REGIS_M];
        float regisB[2][REGIS_N];
        float regisC[REGIS_M][REGIS_N] = {0};

        const int threadDimX = BLOCK_N / REGIS_N;
        const int threadDimY = BLOCK_M / REGIS_M;
        const int threadCount = threadDimX * threadDimY;
        const int tid = threadIdx.y * threadDimX + threadIdx.x;

        ///register for buffering elements during transporting global to shared mem
        float bufferA[BLOCK_M * BLOCK_K / threadCount] = {0};
        float bufferB[BLOCK_N * BLOCK_K / threadCount] = {0};

        ///prepare configs for reading global
        float* ptrA = A->elements;
        float* ptrB = B->elements;
        const int blockM = blockIdx.y * BLOCK_M;
        const int blockN = blockIdx.x * BLOCK_N;

        const int readThreadPerRowA = BLOCK_K;
        const int readThreadPerRowB = BLOCK_N;

        //the location each thread should be reading relative to smem
        const int readRowA = tid / readThreadPerRowA;
        const int readColA = tid % readThreadPerRowA;

        const int readRowB = tid / readThreadPerRowB;
        const int readColB = tid % readThreadPerRowB;

        //these values are used to determine the amount of rows to jump
        //if there is the need to do read multiple times
        const int readRowStrideA = threadCount / readThreadPerRowA;
        const int readRowStrideB = threadCount / readThreadPerRowB;

        #pragma unroll
        for (int i = 0; i < BLOCK_M; i += readRowStrideA) {
            if (blockM + readRowA + i < M && readColA < K) {
                int procReadNDim = readColA / (FH * FW);
                int procReadCol = readColA % (FH * FW);
                tileA[0][readColA][readRowA + i] = ptrA[procReadNDim * FH * FW * FC +
                                     (blockM + readRowA + i) * FH * FW + procReadCol];
            } else {
                tileA[0][readColA][readRowA + i] = 0;
            }
        }

        ///this section is modified from its original state to suit the need for implicit gemm
        ///we are using A special mapping to instantiate patches as trajectories of conv filters
        #pragma unroll
        for (int i = 0; i < BLOCK_K; i += readRowStrideB) {
            if (readRowB + i < K && blockN + readColB < N) {

                //map buffer matrix cords to the 3 dimensional feature cords
                int ic = (readColB + blockN) / (OH * OW);
                int oh = ((readColB + blockN) % (OH * OW)) / OW;
                int ow = ((readColB + blockN) % (OH * OW)) % OW;

                int in = (readRowB + i) / (FH * FW);
                int rst = (readRowB + i) % (FH * FW);
                int fh = ((rst) % (int)(FH * FW)) / (int)FW;
                int fw = ((rst) % (int)(FH * FW)) % (int)FW;
                int ih = oh * strideH - padH + fh * strideH;
                int iw = ow * strideW - padW + fw * strideW;
                //do memory access
                tileB[0][readRowB + i][readColB] = ih >= 0 && iw >= 0 && ih < IH && iw < IW ?
                                                   ptrB[in * IC * IH * IW + ic * IH * IW + ih * IW + iw] : 0;
            } else {
                tileB[0][readRowB + i][readColB] = 0;
            }
        }
        __syncthreads();


        #pragma unroll
        for (int rm = 0; rm < REGIS_M; rm += 4) {
            toFloat4R(regisA[0][rm]) = toFloat4R(tileA[0][0][REGIS_M * threadIdx.y + rm]);
        }

        #pragma unroll
        for (int rn = 0; rn < REGIS_N; rn += 4) {
            toFloat4R(regisB[0][rn]) = toFloat4R(tileB[0][0][REGIS_N * threadIdx.x + rn]);
        }

        ///main loop
        int writeStageFlag = 1;
        #pragma unroll
        for (int nextTileID = BLOCK_K; nextTileID < K + BLOCK_K; nextTileID += BLOCK_K) {
            //prefetch
            if (nextTileID < K) {

                #pragma unroll
                for (int i = 0; i < BLOCK_M; i += readRowStrideA) {
                    int loadIndex = i / readRowStrideA;
                    int procReadNDim = (readColA + nextTileID) / (FH * FW);
                    int procReadCol = (readColA + nextTileID) % (FH * FW);
                    bufferA[loadIndex] = blockM + readRowA + i < M && readColA + nextTileID < K ?
                                         ptrA[procReadNDim * FH * FW * FC +
                                                 (blockM + readRowA + i) * FH * FW + procReadCol] : 0;
                }

                #pragma unroll
                for (int i = 0; i < BLOCK_K; i += readRowStrideB) {

                    //calculate remapping
                    int loadIndex = i / readRowStrideB;
                    int ic = (readColB + blockN) / (OH * OW);
                    int oh = ((readColB + blockN) % (OH * OW)) / OW;
                    int ow = ((readColB + blockN) % (OH * OW)) % OW;

                    int in = (readRowB + i + nextTileID) / (FH * FW);
                    int rst = (readRowB + i + nextTileID) % (FH * FW);
                    int fh = (rst % (int)(FH * FW)) / (int)FW;
                    int fw = (rst % (int)(FH * FW)) % (int)FW;
                    int ih = oh * strideH - padH + fh * strideH;
                    int iw = ow * strideW - padW + fw * strideW;

                    //do memory access
                    bufferB[loadIndex] =
                            (readRowB + i + nextTileID < K && blockN + readColB < N) && (ih >= 0 && iw >= 0)
                            && (ih < IH && iw < IW) ? ptrB[in * IC * IH * IW + ic * IH * IW + ih * IW + iw] : 0;
                }
            }

            int nextStageFlag = writeStageFlag ^ 1;

            //compute the part that is already in the registers and load the next segment
            #pragma unroll
            for (int i = 0; i < BLOCK_K - 1; i++) {

                #pragma unroll
                for (int rm = 0; rm < REGIS_M; rm += 4) {
                    toFloat4R(regisA[(i + 1) % 2][rm]) = toFloat4R(
                            tileA[nextStageFlag][i + 1][REGIS_M * threadIdx.y + rm]);
                }

                #pragma unroll
                for (int rn = 0; rn < REGIS_N; rn += 4) {
                    toFloat4R(regisB[(i + 1) % 2][rn]) = toFloat4R(
                            tileB[nextStageFlag][i + 1][REGIS_N * threadIdx.x + rn]);
                }

                #pragma unroll
                for (int rm = 0; rm < REGIS_M; rm++) {
                    #pragma unroll
                    for (int rn = 0; rn < REGIS_N; rn++) {
                        regisC[rm][rn] += regisA[i % 2][rm] * regisB[i % 2][rn];
                    }
                }
            }

            //load the data in the register buffers to tiles
            if (nextTileID < K) {
                #pragma unroll
                for (int i = 0; i < BLOCK_M; i += readRowStrideA) {
                    int loadIndex = i / readRowStrideA;
                    tileA[writeStageFlag][readColA][readRowA + i] = bufferA[loadIndex];
                }

                #pragma unroll
                for (int i = 0; i < BLOCK_K; i += readRowStrideB) {
                    int loadIndex = i / readRowStrideB;
                    tileB[writeStageFlag][readRowB + i][readColB] = bufferB[loadIndex];
                }

                __syncthreads();
                writeStageFlag ^= 1;  //switch
            }
            #pragma unroll
            for (int rm = 0; rm < REGIS_M; rm += 4) {
                toFloat4R(regisA[0][rm]) = toFloat4R(
                        tileA[nextStageFlag ^ 1][0][REGIS_M * threadIdx.y + rm]);
            }

            #pragma unroll
            for (int rn = 0; rn < REGIS_N; rn += 4) {
                toFloat4R(regisB[0][rn]) = toFloat4R(
                        tileB[nextStageFlag ^ 1][0][REGIS_N * threadIdx.x + rn]);
            }

            #pragma unroll
            for (int rm = 0; rm < REGIS_M; rm++) {
                #pragma unroll
                for (int rn = 0; rn < REGIS_N; rn++) {
                    regisC[rm][rn] += regisA[1][rm] * regisB[1][rn];
                }
            }
        }

        #pragma unroll
        for(int rm = 0; rm < REGIS_M; rm ++){
            #pragma unroll
            for(int rn = 0; rn < REGIS_N; rn ++){
                if((blockM + threadIdx.y * REGIS_M + rm < M && blockN + threadIdx.x * REGIS_N + rn < N)) {
                    C->elements[(blockM + threadIdx.y * REGIS_M + rm) * N
                                + (blockN + threadIdx.x * REGIS_N + rn)] += regisC[rm][rn];
                }
            }
        }
    }
    
    __global__ void convBiasD(Tensor* Y, Tensor* bias){
        uint32 tid = threadIdx.x + blockIdx.x * blockDim.x;
        uint32 ndim = blockIdx.y;
        uint32 globalWarpID = tid / WARP_SIZE;
        uint32 laneID = threadIdx.x % WARP_SIZE;
        
        //each warp will be covering 1 channel from the input tensor
        uint32 readRange = Y->dims.h * Y->dims.w;
        uint32 readBeg = laneID * (readRange / WARP_SIZE);
        uint32 readEnd = laneID == WARP_SIZE - 1 ? readRange : readBeg + readRange / WARP_SIZE;
        if(globalWarpID >= bias->dims.size) return;
    
        float biasVal = bias->elements[globalWarpID];
        #pragma unroll
        for(uint32 i = readBeg; i < readEnd; i ++){
            float val = Y->elements[ndim * (Y->dims.size / Y->dims.n) +
                                    globalWarpID * readRange + i];
            val = val + biasVal;
            Y->elements[ndim * (Y->dims.size / Y->dims.n) + globalWarpID * readRange + i] = val;
        }
    }
    
    void assertConv(Tensor* filters, Tensor* features, Tensor* featureOut, uint32 strideH, uint32 strideW, uint32 padH, uint32 padW){
        if(featureOut->dims.h != (features->dims.h - filters->dims.h + 2 * padH) / strideH + 1){
            logFatal(seio::LOG_SEG_SEBLAS, "Tensor assert failed:");
            logFatal(seio::LOG_SEG_SEBLAS, "assertConv: rows (h) relationship did not satisfy");
            throw std::invalid_argument("assertConv: rows (h) relationship did not satisfy");
        }
        
        if(featureOut->dims.w != (features->dims.w - filters->dims.w + 2 * padW) / strideW + 1){
            logFatal(seio::LOG_SEG_SEBLAS, "Tensor assert failed:");
            logFatal(seio::LOG_SEG_SEBLAS, "assertConv: cols (w) relationship did not satisfy");
            throw std::invalid_argument("assertConv: cols (w) relationship did not satisfy");
        }
        
        if(featureOut->dims.c != filters->dims.n || features->dims.c != filters->dims.c){
            logFatal(seio::LOG_SEG_SEBLAS, "Tensor assert failed:");
            logFatal(seio::LOG_SEG_SEBLAS, "assertConv: channels (c) relationship did not satisfy");
            throw std::invalid_argument("assertConv: channels relationship did not satisfy");
        }
        
        if(featureOut->dims.n != features->dims.n){
            logFatal(seio::LOG_SEG_SEBLAS, "Tensor assert failed:");
            logFatal(seio::LOG_SEG_SEBLAS, "assertConv: batch (n) relationship did not satisfy");
            throw std::invalid_argument("assertConv: batch relationship did not satisfy");
        }
    }
    
    Tensor* conv(Tensor *A, Tensor *B, Tensor *C, int strideH, int strideW, int padH, int padW, Tensor* biases) {
        assertConv(A,B,C, strideH, strideW, padH, padW);
        
        uint32 M = A->dims.n;
        uint32 N = C->dims.h * C->dims.w * C->dims.n;
        
        dim3 grid = dim3((N + BN - 1) / BN, (M + BM - 1) / BM);
        dim3 block = dim3(BN / RN, BM / RM);
        
        gemmImplicit4D<BM, BN, BK, RM, RN><<<grid, block>>>(A, B, C, strideH, strideW, padH, padW, biases);
        hipDeviceSynchronize();
        assertCuda(__FILE__, __LINE__);
        return C;
    }
    
    
    //C is the errors of prev layer and B is for this layer
    Tensor* convDerive(Tensor *A, Tensor *B, Tensor *C, int strideH, int strideW, int padH, int padW) {
        assertConv(A, C, B, strideH, strideW, padH, padW);
        
        uint32 M = A->dims.c;
        uint32 N = C->dims.h * C->dims.w * C->dims.n;
        
        int newPadH = (int)(C->dims.h + A->dims.h - 1 - B->dims.h * strideH)/2;
        int newPadW = (int)(C->dims.w + A->dims.w - 1 - B->dims.w * strideH)/2;
        
        dim3 grid = dim3((N + BN - 1) / BN, (M + BM - 1) / BM);
        dim3 block = dim3(BN / RN, BM / RM);
        
        gemmImplicitBackprop<BM, BN, BK, RM, RN><<<grid, block>>>(A, B, C, strideH, strideW, newPadH, newPadW);
        hipDeviceSynchronize();
        assertCuda(__FILE__, __LINE__);
        return C;
    }
    
    //A is the error of layer, B is the input, C is the filters grads
    Tensor* convError(Tensor *A, Tensor *B, Tensor *C, int strideH, int strideW, int padH, int padW) {
        assertConv(C, B, A, strideH, strideW, padH, padW);
        
        uint32 M = A->dims.c;
        uint32 N = C->dims.h * C->dims.w * C->dims.c;
        
        dim3 grid = dim3((N + BN - 1) / BN, (M + BM - 1) / BM);
        dim3 block = dim3(BN / RN, BM / RM);
        
        gemmImplicitError<BM, BN, BK, RM, RN><<<grid, block>>>(A, B, C, strideH, strideW, padH, padW);
        hipDeviceSynchronize();
        assertCuda(__FILE__, __LINE__);
        return A;
    }
    
    Tensor* convBias(Tensor* Y, Tensor* bias){
        assert(bias->dims.size == Y->dims.c);
        uint32 warps = (CUDA_BLOCK_SIZE.x * CUDA_BLOCK_SIZE.y) / WARP_SIZE;
        uint32 block = CUDA_BLOCK_SIZE.x * CUDA_BLOCK_SIZE.y;
        dim3 grid = dim3((bias->dims.size + warps - 1) / warps,
                         Y->dims.n);
        convBiasD<<<grid, block>>>(Y, bias);
        hipDeviceSynchronize();
        assertCuda(__FILE__, __LINE__);
        return Y;
    }
}
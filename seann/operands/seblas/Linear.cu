#include "hip/hip_runtime.h"
//
// Created by Dylan on 6/20/2022.
//

#include "Linear.cuh"

namespace seann {
    void Linear::initNetParams(OptimizerInfo *info, shape4 inShape) {
        INPUT_SIZE = inShape.size/inShape.n;
        X = Parameter::declare(inShape.n , 1, INPUT_SIZE, 1);
        Y = Parameter::create(inShape.n, 1, OUTPUT_SIZE, 1);
        weights = (new NetParam(info, OUTPUT_SIZE, INPUT_SIZE))->setWeight();
        biases = new NetParam(info, OUTPUT_SIZE, 1);
    }
    
    // a[l] = w[l] * a[l-1] + b[l]
    void Linear::forward() {
        linear(X->A, weights->data(), biases->data(), Y->A);
    }
    
    void Linear::paramGrads() {
        linearParamGrad(Y->dA, X->A, weights->grad(), biases->grad());
    }
    
    void Linear::updateParams() {
        weights->opt->apply();
        biases->opt->apply();
    }
    
    void Linear::batchUpdateParams() {
        weights->opt->batchApply();
        biases->opt->batchApply();
    }
    
    void Linear::xGrads() {
        // ∂x = w^T * ∂z
        *Y->dA + Y->dAReserve;
        linearXGrad(Y->dA, weights->data(), X->dA);
        Y->dAReserve->constFill(0);
    }
    
    void Linear::randFillNetParams() {
        uint32 K = weights->data()->dims.w;
        weights->data()->randNormal( 0, (float)sqrt(2.0 / (float) K));
        biases->data()->randNormal(0, (float)sqrt(2.0 / (float) biases->data()->dims.size));
    }
    
    void Linear::zeroGrads() {
        weights->opt->zeroGrad();
        biases->opt->zeroGrad();
    }
    
    float Linear::getOptimLR() {
        return weights->opt->LEARNING_RATE;
    }
    
    void Linear::updateOptimLR(float val) {
        weights->opt->LEARNING_RATE = val;
        biases->opt->LEARNING_RATE = val;
    }
} // seann
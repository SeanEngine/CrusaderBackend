//
// Created by Dylan on 6/29/2022.
//

#include "ShortcutEndpoint.cuh"

namespace seann {
    void ShortcutEndpoint::initNetParams(OptimizerInfo *info, shape4 inShape) {
        
        X = Parameter::declare(inShape);
        Y = Parameter::create(inShape);
    }
    
    void ShortcutEndpoint::postWaiveInit(OptimizerInfo* inf) {
        if(isContainer) {
            //trace the other endpoint
            OperandBase* tracer = this->tracePrev();
            while(tracer->OPERAND_ID() != OPR_CTRL_SHORTCUT_SRC ||
                    ((ShortcutEndpoint*)tracer)->uuid != uuid) {
                tracer = tracer->tracePrev();
                assert(tracer != nullptr);
            }
            other = (ShortcutEndpoint*)tracer;
            assert(!other->isContainer);
            
            if(operandCount > 0) {
                branchOperands[0]->initNetParams(inf, other->Y->A->dims);
                for (auto i = 1; i < operandCount; i++) {
                    branchOperands[i]->initNetParams(inf, branchOperands[i - 1]->Y->A->dims);
                }
                assert(branchOperands[operandCount - 1]->Y->A->dims == Y->A->dims);
    
                branchOperands[0]->bindPrev(other);
                branchOperands[0]->bindInput(other->Y);
                branchOperands[0]->X->dA = other->Y->dAReserve;
    
                for (auto i = 1; i < operandCount; i++) {
                    branchOperands[i]->bindPrev(branchOperands[i - 1]);
                    branchOperands[i]->bindInput(branchOperands[i - 1]->Y);
                }
    
                for (auto i = 0; i < operandCount; i++) {
                    branchOperands[i]->postWaiveInit(inf);
                }
            }
        }else{
            OperandBase* tracer = this->traceNext();
            while(tracer->OPERAND_ID() != OPR_CTRL_SHORTCUT_CTN ||
                  ((ShortcutEndpoint*)tracer)->uuid != uuid) {
                tracer = tracer->traceNext();
                assert(tracer != nullptr);
            }
            other = (ShortcutEndpoint*)tracer;
            assert(other->isContainer);
        }
    }
    
    void ShortcutEndpoint::forward() {
        //transport the main branch
        hipMemcpy(Y->A->elements, X->A->elements,
                   X->A->dims.size * sizeof(float), hipMemcpyDeviceToDevice);
        assertCuda(__FILE__, __LINE__);
        if(isContainer) {
            //branch
            if(operandCount > 0) {
                for (auto i = 0; i < operandCount; i++) {
                    branchOperands[i]->forward();
                }
                *Y->A + branchOperands[operandCount - 1]->Y->A;
            }else{
                //Identity shortcut
                *Y->A + other->Y->A;
            }
        }
    }
    
    void ShortcutEndpoint::xGrads() {
        if(isContainer) {
            if(operandCount > 0) {
                hipMemcpy(branchOperands[operandCount-1]->Y->dA->elements, Y->dA->elements,
                           Y->A->dims.size * sizeof(float), hipMemcpyDeviceToDevice);
                assertCuda(__FILE__, __LINE__);
    
                for (auto i = 0; i < operandCount; i++) {
                    branchOperands[i]->xGrads();
                }
            }else{
                //Identity shortcut
                hipMemcpy(other->Y->dAReserve->elements, Y->dA->elements,
                           Y->A->dims.size * sizeof(float), hipMemcpyDeviceToDevice);
                assertCuda(__FILE__, __LINE__);
            }
        }else{
            *Y->dA + Y->dAReserve;
            Y->dAReserve->constFill(0);
        }
        hipMemcpy(X->dA->elements, Y->dA->elements,
                   X->A->dims.size * sizeof(float), hipMemcpyDeviceToDevice);
        assertCuda(__FILE__, __LINE__);
    }
    
    void ShortcutEndpoint::paramGrads() {
        if(isContainer){
            if(operandCount > 0) {
                for (auto i = 0; i < operandCount; i++) {
                    branchOperands[i]->paramGrads();
                }
            }
        }
    }
    
    void ShortcutEndpoint::updateParams() {
        if(isContainer){
            if(operandCount > 0) {
                for (auto i = 0; i < operandCount; i++) {
                    branchOperands[i]->updateParams();
                }
            }
        }
    }
    
    void ShortcutEndpoint::batchUpdateParams() {
        if(isContainer){
            if(operandCount > 0) {
                for (auto i = 0; i < operandCount; i++) {
                    branchOperands[i]->batchUpdateParams();
                }
            }
        }
    }
    
    void ShortcutEndpoint::randFillNetParams() {
        if(isContainer) {
            if(operandCount > 0) {
                for (auto i = 0; i < operandCount; i++) {
                    branchOperands[i]->randFillNetParams();
                }
            }
        }
    }
    
    void ShortcutEndpoint::zeroGrads() {
        if(isContainer) {
            if(operandCount > 0) {
                for (auto i = 0; i < operandCount; i++) {
                    branchOperands[i]->zeroGrads();
                }
            }
        }
    }
} // seann
//
// Created by Dylan on 7/4/2022.
//

#include "ChannelConcatenater.cuh"
#include "../../../seblas/assist/Inspections.cuh"

namespace seann {
    void ChannelConcatenater::forward() {
        concat(Xs, paramCount, Y);
    }
    
    void ChannelConcatenater::xGrads() {
        *Y->dA + Y->dAReserve;
        concatGrads(Y, Xs, paramCount);
        hipMemcpy(Xs[0]->dA->elements, Xs[0]->dAReserve->elements,
                   Xs[0]->dA->dims.size * sizeof(float), hipMemcpyDeviceToDevice);
        Y->dAReserve->constFill(0);
        Xs[0]->dAReserve->constFill(0);
    }
    
    uint32 ChannelConcatenater::encodeInfo(fstream *fout, uint64 offset) {
        fout->seekp((long long) offset);
        fout->write((char*)&paramCount, sizeof(uint32));
        fout->write((char*)&outputChannels, sizeof(uint32));
        if (paramCount > 1) {
            fout->write((char *) locations, sizeof(uint32) * (paramCount - 1));
            return sizeof(uint32) * (2 + paramCount - 1);
        }
        return sizeof(uint32) * 2;
    }
    
    uint32 ChannelConcatenater::encodeNetParams(fstream *fout, uint64 offset) {
        return 0;
    }
} // seann
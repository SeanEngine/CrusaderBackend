#include "hip/hip_runtime.h"
//
// Created by Dylan on 6/25/2022.
//

#include "cuConv2D.cuh"

namespace seann {
    void cuConv2D::initNetParams(OptimizerInfo *info, shape4 inShape) {
        filter = (new NetParam(info, filterShape))->setWeight();
        if (WITH_BIAS) bias = new NetParam(info, filterShape.n, 1);
        X = Parameter::declare(inShape); //input features
        shape4 outShape = {
                X->A->dims.n,
                filterShape.n,
                (inShape.h + 2 * padH - filterShape.h) / strideH + 1,
                (inShape.w + 2 * padW - filterShape.w) / strideW + 1};
    
        Y = Parameter::create(outShape);
    }
    
    void cuConv2D::forward() {
        float alpha = 1.0f, beta = 0.0f;
        hipdnnConvolutionForward(
                cudnn,
                &alpha,
                X->A->cudnnDesc, X->A->elements,
                filterDesc, filter->data()->elements,
                 convDesc,
                HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                nullptr, 0,
                &beta,
                Y->A->cudnnDesc, Y->A->elements);
    }
    
    void cuConv2D::xGrads() {
        float alpha = 1.0f, beta = 0.0f;
        *Y->dA + Y->dAReserve;
        hipdnnConvolutionBackwardData(
                cudnn,
                &alpha,
                filterDesc, filter->data()->elements,
                Y->dA->cudnnDesc, Y->dA->elements,
                convDesc,
                HIPDNN_CONVOLUTION_BWD_DATA_ALGO_0,
                nullptr, 0,
                &beta,
                X->dA->cudnnDesc, X->dA->elements);
        Y->dAReserve->constFill(0);
    }
    
    void cuConv2D::paramGrads() {
        float alpha = 1.0f, beta = 0.0f;
        hipdnnConvolutionBackwardFilter(
                cudnn,
                &alpha,
                X->A->cudnnDesc, X->A->elements,
                Y->dA->cudnnDesc, Y->dA->elements,
                convDesc,
                HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_0,
                nullptr, 0,
                &beta,
                filterDesc, filter->grad()->elements);
        
        if(WITH_BIAS){
            hipdnnConvolutionBackwardBias(
                    cudnn,
                    &alpha,
                    Y->dA->cudnnDesc, Y->dA->elements,
                    &beta,
                    bias->grad()->cudnnDesc, bias->grad()->elements
                    );
        }
    }
    
    void cuConv2D::updateParams() {
        filter->opt->apply();
        if(WITH_BIAS) bias->opt->apply();
    }
    
    void cuConv2D::batchUpdateParams() {
        filter->opt->batchApply();
        if(WITH_BIAS) bias->opt->batchApply();
    }
    
    void cuConv2D::randFillNetParams() {
        uint32 K = filter->data()->dims.size / filter->data()->dims.n;
        filter->data()->randNormal(0, (float)sqrt(2.0 / (float) K));
        if (WITH_BIAS)
            bias->data()->randNormal(0, (float)sqrt(2.0 / (float)filter->data()->dims.n));
    }
    
    void cuConv2D::zeroGrads() {
        filter->opt->zeroGrad();
        if (WITH_BIAS) bias->opt->zeroGrad();
    }
    
} // seann
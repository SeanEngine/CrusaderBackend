#include "validation/UnitTestTools.cuh"
#include "seblas/operations/cuOperations.cuh"
#include "hipDNN.h"
#include "seann/seann.cuh"
#include "seio/data/Dataset.cuh"
#include "seio/data/DataLoader.cuh"


using namespace std::this_thread;
using namespace chrono;
using namespace seblas;
using namespace seann;

int main(int argc, char** argv) {

    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);
    auto *model = new Sequential(shape4(64, 3, 32, 32), {

            new cuConv2D(cudnn,shape4(32, 3, 3, 3), 1, 1, 1, 1, false),
            new BatchNorm(),
            new ReLU(),
            new cuConv2D(cudnn,shape4(32, 32, 3, 3), 1, 1, 1, 1, false),
            new BatchNorm(),
            new ReLU(),
            new MaxPool2D(2, 2),
            new Dropout(0.8),

            new cuConv2D(cudnn,shape4(64, 32, 3, 3), 1, 1, 1, 1, false),
            new BatchNorm(),
            new ReLU(),
            new cuConv2D(cudnn,shape4(64, 64, 3, 3), 1, 1, 1, 1, false),
            new BatchNorm(),
            new ReLU(),
            new MaxPool2D(2, 2),
            new Dropout(0.7),

            new cuConv2D(cudnn,shape4(128, 64, 3, 3), 1, 1, 1, 1, false),
            new BatchNorm(),
            new ReLU(),
            new cuConv2D(cudnn,shape4(128, 128, 3, 3), 1, 1, 1, 1, false),
            new BatchNorm(),
            new ReLU(),
            new MaxPool2D(2, 2),
            new Dropout(0.6),

            new Linear(128),
            new BatchNorm(),
            new ReLU(),
            new Dropout(0.5),

            new Linear(10),
            new Softmax()
    });

    OptimizerInfo *info = new OPTIMIZER_MOMENTUM(0.001);

    model->construct(info);
    model->randInit();
    model->setLoss(crossEntropyLoss, crossEntropyCalc);

    auto *dataset = Dataset::construct(6400, 64, 50000, 60000, 500,
                                       shape4(3, 32, 32), shape4(10, 1));
    const char *BASE_PATH = R"(D:\Resources\Datasets\cifar-10-bin\data_batch_)";
    for (int i = 0; i < 5; i++) {
        string binPath = BASE_PATH + to_string(i + 1) + ".bin";
        fetchCIFAR(dataset, binPath.c_str(), i);
    }
    const char* TEST_BASE_PATH = R"(D:\Resources\Datasets\cifar-10-bin\test_batch.bin)";
    fetchCIFAR(dataset, TEST_BASE_PATH, 5);

    dataset->allocTestSet(3200);

    model->train(dataset, true, 1);
}